
#include <hip/hip_runtime.h>
__global__ void DepthwiseGEMM(const float *A,const float *B,const int Aheight,const int Awidth,const int Bwidth, const float over_bound,float *C)
{
	float Cvalue = 0;
	float prod_tmp;
	int Bheight = Awidth;
	
	int chn = blockIdx.z;

	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;
	
	for (int e = 0; e < Awidth; ++e){
		prod_tmp = A[chn * Aheight * Awidth + Aheight * e + row]*B[chn*Bheight*Bwidth + col * Bheight+ e];
		if(prod_tmp>over_bound)
			prod_tmp=over_bound;
		Cvalue+=prod_tmp;
		if(Cvalue>over_bound)
			Cvalue=over_bound;
	}	
	C[chn*Aheight*Bwidth + Aheight*col + row] = Cvalue;
}