
#include <hip/hip_runtime.h>
/********************************************************
*	Author: Zhao Mingxin
*	Date:	2018/12/11
*	Description: CUDA Kernel for DepthwiseGEMM. As GPU is good at
*	dealing with 32 bits computation and gets performance degradation
*	when bit-width is longer than 32, so the current DepthwiseGEMM 
*	version can't give right outputs when fixed point is more than 32 bits.
*
*	NOTE:	If you have any issues about this code, please
*	feedback.
*	Homepage:	https://jackgittes.github.io
*********************************************************/
__global__ void DepthwiseGEMM(const int *A,const int *B,const int Aheight,const int Awidth,const int Bwidth, const int up_bound,const int low_bound,int *C)
{
	int Cvalue = 0;
	int prod_tmp;
	int Bheight = Awidth;
	
	int chn = blockIdx.z;
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;
	
	for (int e = 0; e < Awidth; ++e){
		prod_tmp = A[chn * Aheight * Awidth + Aheight * e + row]*B[chn*Bheight*Bwidth + col * Bheight+ e];
		if(prod_tmp>up_bound)
			prod_tmp = up_bound;
		if(prod_tmp<low_bound)
			prod_tmp = low_bound;
		Cvalue += prod_tmp;
		if(Cvalue>up_bound)
			Cvalue=up_bound;
		if(Cvalue<low_bound)
			Cvalue=low_bound;
	}	
	C[chn*Aheight*Bwidth + Aheight*col + row] = Cvalue;
}