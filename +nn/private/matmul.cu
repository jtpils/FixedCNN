
#include <hip/hip_runtime.h>
__global__ void MatMulKernel(const float *A,const float *B,const int Aheight,const int Awidth,const int Bwidth, const float over_bound,float *C)
{
	// Each thread computes one element of C
	// by accumulating results into Cvalue
	float Cvalue = 0;
	float prod_tmp;
	int Bheight = Awidth;
	
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
/* 	for (int e = 0; e < Awidth; ++e)
		Cvalue += A[row * Awidth + e]*B[e * Bwidth + col];
	C[row * Cwidth + col] = Cvalue; */
	
/* 	for (int e = 0; e < Awidth; ++e){
		
		Cvalue += A[Aheight * e + row]*B[col * Bheight + e];
		
	}	
	C[Aheight*col + row] = Cvalue; */
	for (int e = 0; e < Awidth; ++e){
		prod_tmp = A[Aheight * e + row]*B[col * Bheight + e];
		if(prod_tmp>over_bound)
			prod_tmp=over_bound;
		Cvalue+=prod_tmp;
		if(Cvalue>over_bound)
			Cvalue=over_bound;
	}	
	C[Aheight*col + row] = Cvalue;
}