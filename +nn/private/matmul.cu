
#include <hip/hip_runtime.h>
__global__ void MatMulKernel(const int *A,const int *B,const int Aheight,const int Awidth,const int Bwidth, const int up_bound,const int low_bound,int *C)
{
	int Cvalue = 0;
	int prod_tmp;
	int Bheight = Awidth;
	
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	for (int e = 0; e < Awidth; ++e){
		prod_tmp = A[Aheight * e + row]*B[col * Bheight + e];
		if(prod_tmp>up_bound)
			prod_tmp=up_bound;
		if(prod_tmp<low_bound)
			prod_tmp=low_bound;
		
		Cvalue+=prod_tmp;
		if(Cvalue>up_bound)
			Cvalue=up_bound;
		if(Cvalue<low_bound)
			Cvalue=low_bound;
	}	
	C[Aheight*col + row] = Cvalue;
}